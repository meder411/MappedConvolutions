#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include <vector>

#include "cuda_helper.h"
#include "util/cuda/knn.cuh"

namespace mapped_conv {
namespace util {
namespace cuda {

std::vector<torch::Tensor> KNNForward(torch::Tensor ref,  // B x D x N
                                      torch::Tensor query, const int64_t k) {
  const int64_t batch_size    = ref.size(0);
  const int64_t dim           = ref.size(1);
  const int64_t num_ref_pts   = ref.size(2);
  const int64_t num_query_pts = query.size(2);

  torch::Tensor idx = torch::zeros({batch_size, k, num_query_pts},
                                   ref.options().dtype(torch::kLong));
  torch::Tensor dist =
      torch::zeros({batch_size, k, num_query_pts}, ref.options());

  for (int i = 0; i < batch_size; i++) {
    KNNLauncher(ref[i], num_ref_pts, query[i], num_query_pts, dim, k, dist[i],
                idx[i], at::cuda::getCurrentCUDAStream());
    CUDA_CHECK(hipGetLastError());
  }

  return {idx, dist};
}

}  // namespace cuda
}  // namespace util
}  // namespace mapped_conv