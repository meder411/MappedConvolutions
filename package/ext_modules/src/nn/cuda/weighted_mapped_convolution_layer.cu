#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#include "nn/cuda/mapped_im2col.cuh"

namespace mapped_conv {
namespace nn {
namespace cuda {

torch::Tensor WeightedMappedConvForward(
    torch::Tensor input, torch::Tensor sample_map,
    torch::Tensor interp_weights, torch::Tensor weight, torch::Tensor bias,
    int64_t kernel_size, int64_t interpolation) {
  // Useful dimensions to have
  const int64_t nOutputPlanes  = weight.size(0);
  const int64_t nInputPlanes   = weight.size(1);
  const int64_t inputHeight    = input.size(2);
  const int64_t inputWidth     = input.size(3);
  const int64_t outputHeight   = sample_map.size(0);
  const int64_t outputWidth    = sample_map.size(1);
  const int64_t num_interp_pts = interp_weights.size(3);
  const int64_t batchSize      = input.size(0);

  // Initialize output and temporary columns
  torch::Tensor output =
      torch::zeros({batchSize, nOutputPlanes, outputHeight, outputWidth},
                   input.options()) +
      10;
  torch::Tensor columns =
      torch::zeros({kernel_size * nInputPlanes, outputHeight * outputWidth},
                   input.options());

  // For each elt in batch, do:
  const int64_t outputBatchStride = nOutputPlanes * outputHeight * outputWidth;
  for (int64_t b = 0; b < batchSize; b++) {
    // CUDA mapped_im2col
    MappedIm2Col2DWeightedLauncher(input[b], sample_map, interp_weights,
                                   nInputPlanes, inputHeight, inputWidth,
                                   outputWidth, columns.size(1), kernel_size,
                                   interpolation, num_interp_pts, columns);

    // Get cuda stream
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    hipblasSetStream(handle, at::cuda::getCurrentCUDAStream());

    // Call the GEMM function (note that it expects column major matrices)
    const int64_t m = columns.size(1);
    const int64_t k = weight.size(1) * weight.size(2);
    const int64_t n = weight.size(0);
    if (input.dtype() == torch::kDouble) {
      const double alpha = 1.0;
      const double beta  = 0.0;
      hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
                  columns.data<double>(), m, weight.data<double>(), k, &beta,
                  output.data<double>() + b * outputBatchStride, m);
    } else if (input.dtype() == torch::kFloat) {
      const float alpha = 1.0;
      const float beta  = 0.0;
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
                  columns.data<float>(), m, weight.data<float>(), k, &beta,
                  output.data<float>() + b * outputBatchStride, m);
    }
    CUDA_CHECK(hipGetLastError())

    // Use PyTorch to add the bias
    output[b] += bias.view({output[b].size(0), 1, 1});
  }

  return output;
}

torch::Tensor WeightedMappedConvBackwardInput(
    torch::Tensor grad_output, torch::Tensor sample_map,
    torch::Tensor interp_weights, torch::Tensor weight, int64_t inputHeight,
    int64_t inputWidth, int64_t kernel_size, int64_t interpolation) {
  // Useful dimensions to have
  const int64_t nOutputPlanes  = weight.size(0);
  const int64_t nInputPlanes   = weight.size(1);
  const int64_t outputHeight   = grad_output.size(2);
  const int64_t outputWidth    = grad_output.size(3);
  const int64_t num_interp_pts = interp_weights.size(3);
  const int64_t batchSize      = grad_output.size(0);

  // Initialize output and temporary columns
  torch::Tensor input_grad =
      torch::zeros({batchSize, nInputPlanes, inputHeight, inputWidth},
                   grad_output.options());
  torch::Tensor columns =
      torch::zeros({kernel_size * nInputPlanes, outputHeight * outputWidth},
                   grad_output.options());

  // For each elt in batch, do:
  const int64_t outputBatchStride = nOutputPlanes * outputHeight * outputWidth;
  for (int64_t b = 0; b < batchSize; b++) {
    // Get cuda stream
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    hipblasSetStream(handle, at::cuda::getCurrentCUDAStream());

    // Call the GEMM function (note that it expects column major matrices)
    const int64_t m = columns.size(1);
    const int64_t n = weight.size(1) * weight.size(2);
    const int64_t k = weight.size(0);
    if (grad_output.dtype() == torch::kDouble) {
      const double alpha = 1.0;
      const double beta  = 0.0;
      hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha,
                  grad_output.data<double>() + b * outputBatchStride,
                  m,                                  // lda=N
                  weight.data<double>(), n,           // ldb=ck^2
                  &beta, columns.data<double>(), m);  // ldc=N
    } else if (grad_output.dtype() == torch::kFloat) {
      const float alpha = 1.0;
      const float beta  = 0.0;
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha,
                  grad_output.data<float>() + b * outputBatchStride,
                  m,                                 // lda=N
                  weight.data<float>(), n,           // ldb=ck^2
                  &beta, columns.data<float>(), m);  // ldc=N
    }
    CUDA_CHECK(hipGetLastError())

    MappedCol2Im2DWeightedLauncher(
        columns, sample_map, interp_weights, nInputPlanes, inputHeight,
        inputWidth, outputWidth, columns.size(1), kernel_size, interpolation,
        num_interp_pts, input_grad[b]);
  }

  return input_grad;
}

torch::Tensor WeightedMappedConvBackwardWeight(torch::Tensor grad_output,
                                               torch::Tensor sample_map,
                                               torch::Tensor interp_weights,
                                               torch::Tensor input,
                                               int64_t kernel_size,
                                               int64_t interpolation) {
  // Useful dimensions to have
  const int64_t nOutputPlanes  = grad_output.size(1);
  const int64_t nInputPlanes   = input.size(1);
  const int64_t outputHeight   = grad_output.size(2);
  const int64_t outputWidth    = grad_output.size(3);
  const int64_t inputHeight    = input.size(2);
  const int64_t inputWidth     = input.size(3);
  const int64_t num_interp_pts = interp_weights.size(3);
  const int64_t batchSize      = grad_output.size(0);

  // Initialize output and temporary columns
  torch::Tensor weight_grad = torch::zeros(
      {nOutputPlanes, nInputPlanes, kernel_size}, grad_output.options());
  torch::Tensor columns =
      torch::zeros({kernel_size * nInputPlanes, outputHeight * outputWidth},
                   grad_output.options());

  // For each elt in batch, do:
  const int64_t outputBatchStride = nOutputPlanes * outputHeight * outputWidth;
  for (int64_t b = 0; b < batchSize; b++) {
    // Create the column matrix from the input as we would in
    // mapped_conv_forward
    MappedIm2Col2DWeightedLauncher(input[b], sample_map, interp_weights,
                                   nInputPlanes, inputHeight, inputWidth,
                                   outputWidth, columns.size(1), kernel_size,
                                   interpolation, num_interp_pts, columns);

    // Get cuda stream
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    hipblasSetStream(handle, at::cuda::getCurrentCUDAStream());

    // Propagate the gradients from the outputs to the weights using GEMM
    // Note that GEMM expects column major matrices
    const int64_t m = weight_grad.size(1) * weight_grad.size(2);
    const int64_t n = weight_grad.size(0);
    const int64_t k = columns.size(1);
    if (grad_output.dtype() == torch::kDouble) {
      const double alpha = 1.0;
      const double beta  = 1.0;
      hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha,
                  columns.data<double>(), k,
                  grad_output.data<double>() + b * outputBatchStride, k, &beta,
                  weight_grad.data<double>(), m);
    }
    if (grad_output.dtype() == torch::kFloat) {
      const float alpha = 1.0;
      const float beta  = 1.0;
      hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha,
                  columns.data<float>(), k,
                  grad_output.data<float>() + b * outputBatchStride, k, &beta,
                  weight_grad.data<float>(), m);
    }
    CUDA_CHECK(hipGetLastError())
  }

  return weight_grad;
}

}  // namespace cuda
}  // namespace nn
}  // namespace mapped_conv